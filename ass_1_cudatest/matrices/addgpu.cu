#include "hip/hip_runtime.h"
#include "addgpu.h"

__global__ void add(float *a, float *b, float *c, float *e)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    y[i] = a*x[i] + y[i];
}

ADDGPU::ADDGPU(int n_block_, int n_thread_)
    :n_block(n_block_), n_thread(n_thread_)
{
    n = 200;
    k = 500;
    m = 400;
    hipMalloc((void **) &a, n*k*sizeof(float));
    hipMalloc((void **) &b, k*m*sizeof(float));
    hipMalloc((void **) &c, n*m*sizeof(float));
    hipMalloc((void **) &e, n*sizeof(float));
}

void ADDGPU::compute(float* a_, float* b_, float* c_, float* e_, float* d_)
{
    hipMemcpy(x, x_, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y, y_, n*sizeof(float), hipMemcpyHostToDevice);

    axpy<<<n_block,n_thread>>>(a,x,y);

    hipMemcpy(z_, y, n*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(x);
    hipFree(y);
}

