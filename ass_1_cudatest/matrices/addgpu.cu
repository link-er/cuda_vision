#include "hip/hip_runtime.h"
#include "addgpu.h"

__global__ void add(float *a, float *b, float *c, float *e, float *d, int n, int k, int m)
{
    // finding index for the element currently calculated
    int i = threadIdx.x;
    float temp;
    d[i] = 0;
    for(int j=0; j<n; j++){
        temp = 0;
        for(int l=0; l<k; l++){
            temp += a[j*n + l] * b[l*k + i];
        }
        d[i] += c[j*n + i] + e[j];
    }
}

ADDGPU::ADDGPU(int pass_n, int pass_k, int pass_m)
{
    n = pass_n;
    k = pass_k;
    m = pass_m;
    // allocate space for our variables
    hipMalloc((void **) &a1, n*k*sizeof(float));
    hipMalloc((void **) &b1, k*m*sizeof(float));
    hipMalloc((void **) &c1, n*m*sizeof(float));
    hipMalloc((void **) &e1, n*sizeof(float));
    hipMalloc((void **) &d1, m*sizeof(float));
}

void ADDGPU::compute(float** a_, float** b_, float** c_, float* e_, float* d_)
{
    // copy from host to device, to allocated memory
    float* temp_a = new float[n*k];
    for(int i = 0; i < n; ++i)
        for(int j = 0; j < k; ++j)
            temp_a[i*n + j] = a_[i][j];
    hipMemcpy(a1, temp_a, n*k*sizeof(float), hipMemcpyHostToDevice);

    float* temp_b = new float[k*m];
    for(int i = 0; i < k; ++i)
        for(int j = 0; j < m; ++j)
            temp_b[i*k + j] = b_[i][j];
    hipMemcpy(b1, temp_b, k*m*sizeof(float), hipMemcpyHostToDevice);

    float* temp_c = new float[k*m];
    for(int i = 0; i < n; ++i)
        for(int j = 0; j < m; ++j)
            temp_c[i*n + j] = c_[i][j];
    hipMemcpy(c1, temp_c, n*m*sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(e1, e_, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d1, d_, m*sizeof(float), hipMemcpyHostToDevice);

    // call with specifing number of blocks and number of threads
    add<<<1,m>>>(a1, b1, c1, e1, d1, n, k, m);

    // copy the result back to host
    hipMemcpy(d_, d1, m*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(a1);
    hipFree(b1);
    hipFree(c1);
    hipFree(e1);
    hipFree(d1);
    delete [] temp_a;
    delete [] temp_b;
    delete [] temp_c;
}

