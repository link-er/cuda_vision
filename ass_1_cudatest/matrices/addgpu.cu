#include "hip/hip_runtime.h"
#include "addgpu.h"

__global__ void add(float **a, float **b, float **c, float *e, float *d, int n, int k)
{
    // finding index for the element currently calculated
    int i = threadIdx.x;
    float temp;
    d[i] = 0;
    for(int j=0;j<n;j++){
        temp = 0;
        for(int l=0;l<k;l++){
            temp += a[j][l] * b[l][i];
        }
        d[i] += c[j][i] + e[j];
    }
}

ADDGPU::ADDGPU()
{
    n = 200;
    k = 500;
    m = 400;
    // allocate space for our variables
    hipMalloc((void **) &a, n*k*sizeof(float));
    hipMalloc((void **) &b, k*m*sizeof(float));
    hipMalloc((void **) &c, n*m*sizeof(float));
    hipMalloc((void **) &e, n*sizeof(float));
    hipMalloc((void **) &d, m*sizeof(float));
}

void ADDGPU::compute(float** a_, float** b_, float** c_, float* e_, float* d_)
{
    // copy from host to device, to allocated memory
    hipMemcpy(a, a_, n*k*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b, b_, k*m*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c, c_, n*m*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(e, e_, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d, d_, m*sizeof(float), hipMemcpyHostToDevice);

    // call with specifing number of blocks and number of threads
    add<<<1,m>>>(a,b,c,e,d, n, k);

    // copy the result back to host
    hipMemcpy(d_, d, m*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(e);
    hipFree(d);
}

