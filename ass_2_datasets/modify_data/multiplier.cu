#include "hip/hip_runtime.h"
#include "multiplier.h"
#include <iostream>

__global__ void multiplyby2(int n, float *a)
{
    int i = blockIdx.x;
    int j = threadIdx.x;
    a[i*n + j] = 2 * a[i*n + j];
}

MULTIPLIER::MULTIPLIER(int rows, int columns)
{
    m = rows;
    n = columns;
    hipMalloc((void **) &a, m*n*sizeof(float));
}

void MULTIPLIER::compute(float** a_)
{
    float* temp = new float[m*n];
    for(int i=0; i<m; i++)
        for(int j=0; j<n; j++)
            temp[i*n + j] = a_[i][j];

    hipMemcpy(a, temp, m*n*sizeof(float), hipMemcpyHostToDevice);

    multiplyby2<<<m,n>>>(n, a);

    hipMemcpy(temp, a, m*n*sizeof(float), hipMemcpyDeviceToHost);

    for(int i=0; i<m; i++)
        for(int j=0; j<n; j++)
            a_[i][j] = temp[i*n + j];

    hipFree(a);
    delete [] temp;
}

